#include "hip/hip_runtime.h"
#include <cstdio>
#include <cassert>

#include "apsp.h"

#define max_dist 10001
#define batch_size 9
#define alt_batch_size 9
#define type short

__device__ int slice_graph(int n, int *g, int i0, int j0) {
	int i = i0 + threadIdx.y, j = j0 + threadIdx.x;
	return (i < n && j < n) ? g[i * n + j] : max_dist;
}

template <typename T>
__device__ void fill_graph(int n, int *g, int i0, int j0, T ele) {
	int i = i0 + threadIdx.y, j = j0 + threadIdx.x;
	if (i < n && j < n)
		g[i * n + j] = ele;
}

__global__ void first_step(int n, int *g, int r_index) {
	__shared__ int c_sub_graph[32][32];
	int c_i = r_index * 32, c_j = r_index * 32, cur = max_dist;
	c_sub_graph[threadIdx.y][threadIdx.x] = slice_graph(n, g, c_i, c_j);
	__syncthreads();
	for (int k = 0; k < 32; k++)
		cur = min(cur, c_sub_graph[threadIdx.y][k] + c_sub_graph[k][threadIdx.x]);
	fill_graph<int>(n, g, c_i, c_j, cur);
}

__global__ void second_step(int n, int *g, int r_index) {
	__shared__ type c_sub_graph[32][32], v_sub_graph[batch_size][32][32], h_sub_graph[batch_size][32][32];
	int c_i = r_index * 32, c_j = r_index * 32, o = blockIdx.x * batch_size * 32, cur = max_dist;
	c_sub_graph[threadIdx.y][threadIdx.x] = slice_graph(n, g, c_i, c_j);
	for (int p = 0, x = o; p < batch_size; p++, x += 32)
		v_sub_graph[p][threadIdx.y][threadIdx.x] = slice_graph(n, g, x, c_j);
	for (int p = 0, y = o; p < batch_size; p++, y += 32)
		h_sub_graph[p][threadIdx.y][threadIdx.x] = slice_graph(n, g, c_i, y);
	__syncthreads();
	for (int p = 0, x = o; p < batch_size; p++, x += 32) {
		cur = max_dist;
		for (int k = 0; k < 32; k++)
			cur = min(cur, v_sub_graph[p][threadIdx.y][k] + c_sub_graph[k][threadIdx.x]);
		fill_graph<type>(n, g, x, c_j, cur);
	}
	for (int p = 0, y = o; p < batch_size; p++, y += 32) {
		cur = max_dist;
		for (int k = 0; k < 32; k++)
			cur = min(cur, c_sub_graph[threadIdx.y][k] + h_sub_graph[p][k][threadIdx.x]);
		fill_graph<type>(n, g, c_i, y, cur);
	}
}

__global__ void third_step(int n, int *g, int r_index) {
	__shared__ type v_sub_graph[alt_batch_size][32][32], h_sub_graph[alt_batch_size][32][32];
	int cur;
	int c_i = r_index * 32, c_j = r_index * 32;
	int x = blockIdx.y * alt_batch_size * 32, y = blockIdx.x * alt_batch_size * 32;
	for (int k = 0, i = x; k < alt_batch_size; k++, i += 32)
		v_sub_graph[k][threadIdx.y][threadIdx.x] = slice_graph(n, g, i, c_j);
	for (int k = 0, j = y; k < alt_batch_size; k++, j += 32)
		h_sub_graph[k][threadIdx.y][threadIdx.x] = slice_graph(n, g, c_i, j);
	__syncthreads();
	for (int p = 0, i = x; p < alt_batch_size; p++, i += 32) {
		for (int q = 0, j = y; q < alt_batch_size; q++, j += 32) {
			cur = slice_graph(n, g, i, j);
			for (int k = 0; k < 32; k++)
				cur = min(cur, v_sub_graph[p][threadIdx.y][k] + h_sub_graph[q][k][threadIdx.x]);
			fill_graph(n, g, i, j, cur);
		}
	}
}

void apsp(int n, /* device */ int *g) {
	dim3 thr(32, 32);
	const int r_cnt = (n + 32 - 1) / 32;
	const int b_cnt = (n - 1) / (batch_size * 32) + 1;
	const int a_cnt = (n - 1) / (alt_batch_size * 32) + 1;
	for (int r_index = 0; r_index < r_cnt; r_index++) {
		first_step<<<1, thr>>>(n, g, r_index);
		second_step<<<b_cnt, thr>>>(n, g, r_index);
		third_step<<<dim3(a_cnt, a_cnt), thr>>>(n, g, r_index);
	}
}
